#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <Windows.h>
#define PI 3.14159265358979323846
#define M 512//���������
///kernel ����
//���
__global__ void MulVector(hipfftDoubleComplex *a, hipfftDoubleComplex *b, hipfftDoubleComplex *c,int size)
{
	int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����
	int id_tx = threadIdx.x;//���ڿ���̺߳ź��ᣬ�ڼ������뵥Ԫ
	int id_ty = threadIdx.y;//���ڿ���̺߳����ᣬ���ڵڼ�������
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤�ȣ���L
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤�ȣ���N

	int index_real = (id_tx+threadPerBlock_x*id_ty)+id_by*threadPerBlock_x*threadPerBlock_y; //����������ֵ=�ڿ���+����Ŀ���
	int index_ht = id_tx;//���ݺ���������
	int index_echo = index_real;// �ز�����������
	if (index_real < size)
	{
		c[index_real].x = a[index_ht].x * b[index_echo].x - a[index_ht].y * b[index_echo].y;//blockDim.x;//index_ht;//
		c[index_real].y = a[index_ht].x * b[index_echo].y + a[index_ht].y * b[index_echo].x;//blockDim.y;//index_real;//
	}
}
//x.*ww
__global__ void MulVector_xw(hipfftDoubleComplex *a, hipfftDoubleComplex *b, hipfftDoubleComplex *c,int size)
{
	int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����,�ڼ�����ʱ����뵥Ԫ
	int id_bx = blockIdx.x;//��ĺ����ʾ�ڼ���ģ������
	int id_tx = threadIdx.x;//���ڿ���̺߳ź��ᣬ�ڼ������뵥Ԫ
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤�ȣ���2*M
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤�ȣ���1
	int index_x=id_tx+id_by*threadPerBlock_x;//xҪ��˵�����ֵ
	int index_y=index_x;//y���Ҫ���������ֵ
	int index_w=id_tx+size-1+id_by*threadPerBlock_x;//wҪ��˵�����ֵ
	if (id_tx < size)
	{
		c[index_y].x = a[index_x].x * b[index_w].x - a[index_x].y * b[index_w].y;//blockDim.x;//index_ht;//
		c[index_y].y = a[index_x].x * b[index_w].y + a[index_x].y * b[index_w].x;//blockDim.y;//index_real;//
	}
}
//���
__global__ void ChuVector(hipfftDoubleComplex *a, int size,int L)
{
    int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����
	int id_tx = threadIdx.x;//���ڿ���̺߳ź��ᣬ�ڼ������뵥Ԫ
	int id_ty = threadIdx.y;//���ڿ���̺߳����ᣬ���ڵڼ�������
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤�ȣ���L
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤�ȣ���N
	int index_real = (id_tx+threadPerBlock_x*id_ty)+id_by*threadPerBlock_x*threadPerBlock_y; //����������ֵ=�ڿ���+����Ŀ���
	if (index_real < size)
	{
		a[index_real].x = a[index_real].x / L;
		a[index_real].y = a[index_real].y / L;
	}
}
//��׼RFT///////////////////////////////////////////////////////
//                        ����ز���              RFT�����     �����ٶȲ������ٶ������ľ��뵥Ԫ���ֵ,�����ظ����,���뵥Ԫ��С������
__global__ void RFT(hipfftDoubleComplex *pc, hipfftDoubleComplex *Gv, double Vi, int L, int SP, double Tr , double delt_R, double lamda,hipfftDoubleComplex *d_DFT, double *d_offset)
{
	hipfftDoubleComplex Sum={0,0};
	int id_bx = blockIdx.x;//���ڵĿ�ź���,�ڼ�����
	int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����
	int id_tx = threadIdx.x;//���ڿ���̺߳ź���,��ʼ���뵥Ԫ
	int id_ty = threadIdx.y;//���ڿ���̺߳�����,���ڵڼ����ٶ�
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤,��ʼ����
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤��,һ��������ѵ��ٶȸ���
	int BlockPerGid_x= gridDim.x;//grid�еĿ��������
	int BlockPerGid_y= gridDim.y;//grid�еĿ��������
	//__shared__ hipfftDoubleComplex Pc_share[M];
	//����������ֵ=      �ڿ���                      +           ����Ŀ���        *           ��Ĵ�С
	int index_real =(id_tx + id_ty*threadPerBlock_x) + (id_bx + id_by*BlockPerGid_x) * (threadPerBlock_x*threadPerBlock_y);
	int V_index = id_ty + threadPerBlock_y * (id_bx + id_by*BlockPerGid_x) ;//�ٶ�����
	double V=Vi*V_index;//���߳��������ٶ�
	int Strat_R=id_tx;//��ʼ���뵥Ԫ
	int maxoffset=floor(V*M*Tr/delt_R+0.5);//�������߶�
	double Sum_x=0;
	double Sum_y=0;
	int   offset=0;
	if (Strat_R-maxoffset>=0 && index_real<SP*M*L && Strat_R<L)//û���������������ͼ���
	{
		//double fd = 2*V/lamda;
		for(int i=0;i<M;++i)
		{
			//offset=floor(V*i*Tr/delt_R+0.5);
			Strat_R=Strat_R-d_offset[i+id_bx*M+id_by*M*M];
			/*
			Sum_x=cos(2*PI*fd*i*Tr);
			Sum_y=sin(2*PI*fd*i*Tr);
			Sum.x+=pc[Strat_R+i*L].x * Sum_x - pc[Strat_R+i*L].y * Sum_y;//i;//
			Sum.y+=pc[Strat_R+i*L].x * Sum_y + pc[Strat_R+i*L].y * Sum_x;//i;//
			*/
			Sum.x+=pc[Strat_R+i*L].x * d_DFT[i+id_bx*M+id_by*M*M].x - pc[Strat_R+i*L].y * d_DFT[i+id_bx*M+id_by*M*M].y;//i;//
			Sum.y+=pc[Strat_R+i*L].x * d_DFT[i+id_bx*M+id_by*M*M].y + pc[Strat_R+i*L].y * d_DFT[i+id_bx*M+id_by*M*M].x;//i;//
			Strat_R=id_tx;
		}
		Gv[index_real]=Sum;
		Sum.x= 0;
		Sum.y= 0;
	}
	else 
	{
		Gv[index_real].x=0;
		Gv[index_real].y=0;
	}
	Sum.x= 0;
	Sum.y= 0;
}
//ת��
__global__ void ChangeVector(hipfftDoubleComplex *a,hipfftDoubleComplex *b,int L)//����ת��
{
	__shared__ hipfftDoubleComplex share_block[600];//�潻���õ�
	//hipfftDoubleComplex temp;//���õ���ʱ���������ڼĴ�������
	int id_tx = threadIdx.x;//���ڿ���̺߳ź���
	int id_bx = blockIdx.x;//���ڵĿ������
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤��
	if(id_tx < L && id_bx < 2 * M)
	{
		int index_out = id_tx + id_bx*threadPerBlock_x; //���������
		share_block[id_tx] = a[index_out];
	}

	__syncthreads();//���Ϊ������߳�ͬ��

	id_tx = threadIdx.x;//���ڿ���̺߳ź���
	id_bx = blockIdx.x;//���ڵĿ������
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤��
	if(id_tx < L && id_bx < 2 * M)
	{
		int index_in = id_tx*(2*M)+id_bx;//����������
		b[index_in] = share_block[id_tx];
	}

	//temp = a[index_out];
	//a[index_out] = a[index_in];
	//a[index_in] = temp;

}
int main()
{
	//��ʱ��������
	LARGE_INTEGER fp_cpu;//cpu��Ƶ
	QueryPerformanceFrequency(&fp_cpu);//��ȡ��Ƶ
		///GPU��Ϣ˵��
	int MTPB = 1024;//��block���֧���߳���
	int MaxThreadBlockSize[3]={1024,1024,64};//���Ŀ����֧꣬��1024*1024*64����
		/*int dev=1;
	hipDeviceProp_t prop;
	hipGetDevice(&dev);
	printf("GPU�ͺ�:%s\n",prop.name);
	printf("��block���֧���߳���:%d\n",prop.maxThreadsPerBlock);*/
	///���������͸�ֵ
	double fc=100e6;//��Ƶ
	double B=4e6;//����
	double Tao=128e-6;//����
	double Fs=2*B;//����Ƶ��
	double Ts=1/Fs;
	//printf("Ts=%0.8f\n",Ts);
	double mu=B/Tao;//��Ƶ��
	double C=3e8;//����
	double delt_R=C/(2*Fs);//����ֱ���
	double R_start=500*delt_R;//��ʼ����
	double lamda=C/fc;//����
	double PRF=500;//�����ظ�����
	double Tr=1/PRF;//�����ظ�ʱ��
	double Vr=1200;//��ʼ�ٶ�
	printf("Vr=%f\n",Vr);
	double a=0;//���ٶ�
	int    L=Tao*Fs;//���뵥Ԫ���������������
	printf("L=%d\n",L);
	int    N=MTPB/L;//�����߳�����ά����
	printf("N=%d\n",N);
	double *t=(double*)malloc(sizeof(double)*L);//��ʱ��
	double *delt_t=(double*)malloc(sizeof(double)*M);//�ӳ�ʱ��


	for(int i=0;i<L;++i)
	{
		t[i]=-Tao/2+Ts*i;	
		//t[i]=Ts*i;	
	   // printf("t[%d]=%0.8f\n",i+1,t[i]);
	}
	
	///��������/////////////////////////////////////////////
	
	hipfftDoubleComplex  *h_echo,*d_echo; //������,�豸�λز�ʱ��
	hipfftDoubleComplex  *h_echo_fft,*d_echo_fft; //������,�豸�λز�Ƶ��

	hipfftDoubleComplex  *h_ht,*d_ht;    //������,�豸����ѹʱ��ϵ��
	hipfftDoubleComplex  *h_ht_fft,*d_ht_fft;    //������,�豸����ѹʱ��ϵ��

	hipfftDoubleComplex  *h_pc,*d_pc;   //������,�豸��ѹʱ����
	hipfftDoubleComplex  *h_pc_fft,*d_pc_fft;//������,�豸��ѹƵ����

	//�����ڴ棬�Դ�////////////////////////
	hipError_t cudaStatus;//״̬��¼
    h_echo=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex )*M*L);//�����˻ز�ʱ��
	h_echo_fft=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex )*M*L);//�����˻ز�Ƶ��

	h_ht=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex )*L);//��������ѹʱ��ϵ��
	h_ht_fft=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex )*L);//��������ѹƵ��ϵ��

	h_pc=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex )*M*L);//��������ѹʱ����
	h_pc_fft=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex )*M*L);//��������ѹƵ����

    cudaStatus=hipMalloc((void**)&d_echo,sizeof(hipfftDoubleComplex )*M*L);//�豸�λز�ʱ�򿪱��Դ�	    
    if (cudaStatus != hipSuccess) {
        printf( "d_echo hipMalloc failed!\n %d \n",cudaStatus);
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_echo_fft,sizeof(hipfftDoubleComplex )*M*L);//�豸�λز�Ƶ�򿪱��Դ�
	if (cudaStatus != hipSuccess) {
        printf( "d_echo_fft hipMalloc failed!\n");
		return 1;
    }

	cudaStatus=hipMalloc((void**)&d_pc_fft,sizeof(hipfftDoubleComplex )*M*L);//�豸�λز���ѹƵ��
	if (cudaStatus != hipSuccess) {
        printf( "d_pc_fft hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_pc,sizeof(hipfftDoubleComplex )*M*L);//�豸�λز���ѹʱ��
	if (cudaStatus != hipSuccess) {
        printf( "d_pc hipMalloc failed!\n");
		return 1;
    }

	cudaStatus=hipMalloc((void**)&d_ht,sizeof(hipfftDoubleComplex )*L);//�豸�λز���ѹʱ��ϵ���Դ�
	
	if (cudaStatus != hipSuccess) {
        printf( "d_ht hipMalloc failed! \n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_ht_fft,sizeof(hipfftDoubleComplex )*L);//�豸�λز���ѹƵ��ϵ���Դ�
	if (cudaStatus != hipSuccess) {
        printf( "d_ht_fft hipMalloc failed!\n");
		return 1;
    }
	
	///��ʼ��ֵ///////
	///��������ѹʱ��ϵ��
	for(int i=0;i<=L-1;++i)///
	{
		h_ht[i].x=cos(2*PI*(mu/2)*t[i]*t[i]);
		h_ht[i].y=sin(2*PI*(mu/2)*t[i]*t[i]);
	}
	//���ݺ�����ת����
	/*hipfftDoubleComplex t_h;//�����õ���ʱ����
	for (int i = 0; i < L / 2; ++i)//���ݺ���
	{
		if (L - i - 1 >= 0)
		{
			t_h.x = h_ht[i].x;
			h_ht[i].x = h_ht[L - i - 1].x;
			h_ht[L - i - 1].x = t_h.x;
			t_h.y = -1 * h_ht[i].y;//����Ӹ���
			h_ht[i].y = -1 * h_ht[L - i - 1].y;
			h_ht[L - i - 1].y = t_h.y;
		}
	}*/
   //for(int i=0;i<=L-1;++i)printf("h_ht[%d]=%0.8f+%0.8fi\n",i+1,h_ht[i].x,h_ht[i].y);
	///�ز���ֵ
	for(int j=0;j<M;++j)//�ӳټ���
	{
			Vr=Vr+a*Tr*j;
		    delt_t[j]=2*(R_start+Vr*Tr*j)/C;
	}
	for(int i=0;i<M;++i)//��ʱ��
	{
		for(int j=0;j<L; ++j)//��ʱ��
		{
			h_echo[j+i*L].x=cos(2*PI*(mu/2*(t[j]+delt_t[i])*(t[j]+delt_t[i])+fc*delt_t[i]));
			h_echo[j+i*L].y=-sin(2*PI*(mu/2*(t[j]+delt_t[i])*(t[j]+delt_t[i])+fc*delt_t[i]));
			//printf("h_echo[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_echo[j+i*L].x,h_echo[j+i*L].y);
			//if((j+1)%3==0) printf("\n");
		}
		//printf("\n");
	}
	//for(int i=L;i<=2*L-1;++i)printf("h_echo[%d]=%0.8f+%0.8fi\n",i+1,h_echo[i].x,h_echo[i].y);
	//���������ݿ������豸��
	cudaStatus=hipMemcpy(d_echo,h_echo,sizeof(hipfftDoubleComplex )*M*L,hipMemcpyHostToDevice);//�ز����������˵��豸��
	if (cudaStatus != hipSuccess) {
        printf("h_echo->d_echo hipMemcpy failed!\n");
		return 1;
    }
	cudaStatus=hipMemcpy(d_ht,h_ht,sizeof(hipfftDoubleComplex )*L,hipMemcpyHostToDevice);// ��ѹʱ��ϵ��,���������˵��豸��
    if (cudaStatus != hipSuccess) {
        printf("h_ht->d_ht hipMemcpy failed!\n");
		return 1;
    }
	//�Իز�����ѹϵ����fft�任��Ƶ����Ƶ�������ѹ
	//����fft�ƻ�
	LARGE_INTEGER b_pc1,e_pc1,b_pc2,e_pc2;//��ѹ��ʱ
	double time_pc1,time_pc2;
	
	hipfftHandle plan_ML,plan_L;//��L��M����fft����L��1����fft�ƻ�
	hipfftResult Result_fft_ML,Result_fft_L;//���ִ�н��������ֵ���ͻ�����hipError_t�����˹���
	hipfftPlan1d(&plan_ML,L,HIPFFT_Z2Z,M);//���ûز�fft�ƻ�
	QueryPerformanceCounter(&b_pc1);
	Result_fft_ML=hipfftExecZ2Z(plan_ML,d_echo,d_echo_fft,HIPFFT_FORWARD);//�Իز�����FFT
	//��FFT��0��fsǧ����matlab��fftshift������ݱȶ�
	/*if (Result_fft_ML != hipSuccess)
	{
		printf("�ز�fft����!\n");	
		return 1;
	}*/
	/*cudaStatus=hipMemcpy(h_echo_fft,d_echo,sizeof(hipfftDoubleComplex )*M*L,hipMemcpyDeviceToHost);//�ز�Ƶ�������豸�ε�������
	if (cudaStatus != hipSuccess) {
        printf("d_echo_fft->h_echo_fft hipMemcpy failed!\n");
		return 1;
    }*/
	///
	/*for(int i=0;i<M;++i)//��ʱ��
	{
		for(int j=0;j<L; ++j)//��ʱ��
		{
			printf("h_echo_fft[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_echo_fft[j+i*L].x,h_echo_fft[j+i*L].y);
			if((j+1)%2==0) printf("\n");
		}
		printf("\n");
	}*/
	//for(int i=0;i<=L-1;++i)printf("h_echo_fft[%d]=%0.8f+%0.8fi\n",i+1,h_echo_fft[i].x,h_echo_fft[i].y);
	hipfftPlan1d(&plan_L,L,HIPFFT_Z2Z,1);
	Result_fft_L=hipfftExecZ2Z(plan_L,d_ht,d_ht_fft,HIPFFT_FORWARD);//����ѹϵ������FFT HIPFFT_BACKWARD HIPFFT_FORWARD
	if (Result_fft_L != hipSuccess)
	{
		printf("��ѹϵ��fft����!\n");	
		return 1;
	}
	/*cudaStatus=hipMemcpy(h_ht_fft,d_ht_fft,sizeof(hipfftDoubleComplex )*L,hipMemcpyDeviceToHost);//��ѹϵ��Ƶ�������豸�ε�������
	if (cudaStatus != hipSuccess) {
        printf("d_ht_fft->h_ht_fft hipMemcpy failed!\n");
		return 1;
    }
	for(int j=0;j<L; ++j)
	{
			printf("h_ht_fft[%d]=%0.8f+%0.8fi \n ",j+1,h_ht_fft[j].x,h_ht_fft[j].y);
	}*/
	//������ѹ/////////////////////////////////////////////
	//GPU���
	dim3 blcok(1,M);//ÿ������һ�����뵥Ԫ
	dim3 threadPerBlock(L,1);//ÿ�����е�һ���߳���һ������
	MulVector <<<blcok,threadPerBlock>>>(d_ht_fft, d_echo_fft, d_pc_fft, L*M);//����ˣ���֪��Ϊʲô���к���
	//������Ƿ���ȷ/////////////////////
	/*cudaStatus=hipMemcpy(h_pc_fft,d_pc_fft,sizeof(hipfftDoubleComplex )*M*L,hipMemcpyDeviceToHost);//��ѹƵ�������豸�ε�������
    if (cudaStatus != hipSuccess) {
        printf("d_pc_fft->h_pc_fft hipMemcpy failed!\n");
		return 1;
    }*/
	/*for(int i=0;i<M;++i)
	{
		for(int j=0;j<L; ++j)
		{
			printf("h_pc_fft[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_pc_fft[j+i*L].x,h_pc_fft[j+i*L].y);
			printf("\n");
		}
		printf("\n");
	}*/

	Result_fft_ML=hipfftExecZ2Z(plan_ML,d_pc_fft,d_pc,HIPFFT_BACKWARD);//��ѹIFFT
	if (Result_fft_ML != hipSuccess)
	{
		printf("�ز�fft����!\n");	
		return 1;
	}
	ChuVector << <blcok, threadPerBlock >> >(d_pc, M*L,L);//�����
	QueryPerformanceCounter(&e_pc1);
	time_pc1=(double)(e_pc1.QuadPart-b_pc1.QuadPart)/(double)fp_cpu.QuadPart;
	QueryPerformanceCounter(&b_pc2);
	cudaStatus=hipMemcpy(h_pc,d_pc,sizeof(hipfftDoubleComplex )*M*L,hipMemcpyDeviceToHost);//��ѹƵ�������豸�ε�������
	QueryPerformanceCounter(&e_pc2);
	time_pc2=(double)(e_pc2.QuadPart-b_pc2.QuadPart)/(double)fp_cpu.QuadPart;
	printf("��ѹ��ʱ:%f s\n",time_pc1);
	printf("���ݴ�����ʱ:%f s\n",time_pc2);
	printf("����ʱ:%f s\n\n",time_pc1+time_pc2);
    /*if (cudaStatus != hipSuccess) {
        printf("d_pc_fft->h_pc_fft hipMemcpy failed!\n");
		return 1;
    }
	for(int i=33;i<34;++i)
	{
		for(int j=0;j<L; ++j)
		{
			printf("h_pc[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_pc[j+i*L].x,h_pc[j+i*L].y);
			printf("\n");
		}
		printf("\n");
	}*/
	/////////��ѹ�������////////////////////////////////////
	double *h_abs_pc=(double*)malloc(sizeof(double)*M*L);
	for(int i=0; i<M*L; ++i)h_abs_pc[i]=sqrt(h_pc[i].x*h_pc[i].x+h_pc[i].y*h_pc[i].y);
	FILE *fp_pc;
	fp_pc=fopen("d:/Pc.txt","w");
	for(int i=0; i<M; ++i)
	{
		for(int j=0; j<L; ++j)
		{
			fprintf(fp_pc,"%0.8f\t",h_abs_pc[j+i*L]);
		}
		fprintf(fp_pc,"\n");
	}
	//////////////////////////////////////////��ѹ���////////////////////////////////////////////////
	///////////////////////////////////////////RFT///////////////////////////////////////////////////////
	//������������
	double Vb = lamda/2/Tr;//ä��
	//printf("Vb=%f\n",Vb);
	double Vi = Vb/M; //�ٶ���������
	//printf("Vi=%f\n",Vi);
	int    SP =8;//1024*6/M;   //������ä���������//��������һ��ͺ�����GTX650�Կ������Կ������Ƴ���
	printf("SP=%d\n",SP);


	/////����SP�������ٶȡ���Ӧ�Ķ����մ洢�ã��ڼ����Ӧ��DFTϵ���ٴ洢��//////////////
	double *h_fd, *h_offset, *d_offset;
	h_fd=(double *)malloc(sizeof(double )*M*SP);//������fd�洢
	h_offset=(double *)malloc(sizeof(double )*M*SP*M);//������û���ٶ���,ÿ�������ظ����ڵ�ƫ������
	cudaStatus=hipMalloc((void**)&d_offset,sizeof(hipDoubleComplex )*M*SP*M);//�豸��û���ٶ���,ÿ�������ظ����ڵ�ƫ������
	if (cudaStatus != hipSuccess) {
        printf("d_offset hipMalloc failed!\n");
		return 1;
    }
	////����fd,ƫ����//////////
	for(int i=0;i<M*SP;++i)
	{
		h_fd[i] = 2*Vi*i/lamda;
		for(int j=0; j<M; ++j)
		{
			h_offset[i*M+j]=floor(Vi*i*j*Tr/delt_R+0.5);
		}
		
	}
	cudaStatus=hipMemcpy(d_offset,h_offset,sizeof(double)*M*SP*M,hipMemcpyHostToDevice);//��������������ƫ�������豸��
		if (cudaStatus != hipSuccess) {
        printf("h_offset->d_offset  hipMemcpy failed!\n");
		return 1;
    }
	///////////////����DFTϵ��///////////////
	hipDoubleComplex *h_DFT,*d_DFT;
	h_DFT=(hipDoubleComplex *)malloc(sizeof(hipDoubleComplex)*M*SP*M);//������DFT�洢
	cudaStatus=hipMalloc((void**)&d_DFT,sizeof(hipDoubleComplex )*M*SP*M);//�豸��DFT�洢
	for(int i=0; i<M*SP; ++i)
	{
		for(int j=0; j<M; ++j)
		{
			h_DFT[i*M+j].x=cos(2*PI*h_fd[i]*j*Tr);
			h_DFT[i*M+j].y=sin(2*PI*h_fd[i]*j*Tr);
		}	
	}
	cudaStatus=hipMemcpy(d_DFT,h_DFT,sizeof(hipfftDoubleComplex)*M*SP*M,hipMemcpyHostToDevice);//��������������DFTϵ�����豸��
	if (cudaStatus != hipSuccess) {
        printf("h_DFT->d_DFT  hipMemcpy failed!\n");
		return 1;
    }
	free(h_DFT);
	free(h_fd);

	////////////////////////////////////////CPU_RFT//////////////////////////////////////////////////////////
	/*int    Strat_R=0;
	double Sum_x,Sum_y;
	long double offset;
	double2 Sum_cpu;
	hipfftDoubleComplex *Gv_cpu=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*SP*M*L);//V*L,�ٶ�-���뵥Ԫ��ά
	LARGE_INTEGER b_cpu,e_cpu;//��ʱ
	double time_cpu;
	double fd_cpu;
	double *V=(double*)malloc(sizeof(double)*SP*M);
	int    index_real;
	QueryPerformanceCounter(&b_cpu);
	for(int i=0; i<SP*M; ++i)//�ٶ�
	{
		V[i]=Vi*i;
		fd_cpu=2*V[i]/lamda;//������
		//printf("V=%f\t",V[i]);
		//printf("fd_cpu=%f\t",fd_cpu);
		for(int j=0; j<L; ++j)//���뵥Ԫ
		{
			Sum_cpu.x=0;
			Sum_cpu.y=0;
			Strat_R=j;
			//printf("Strat_R=%d\n",Strat_R);
			if(Strat_R -floor(V[i]*Tr*M/delt_R+0.5)>=0 && Strat_R + V[i]*Tr*M/delt_R<L)
			{
				for(int ti=0; ti<M; ++ti)
				{
					offset=floor(V[i]*ti*Tr/delt_R+0.5);
					Strat_R=Strat_R-floor(offset);
					Sum_x=cos(2*PI*fd_cpu*ti*Tr);
					Sum_y=sin(2*PI*fd_cpu*ti*Tr);
					index_real=Strat_R+ti*L;
					Strat_R=j;
					//printf("offset=%d\t index_real=%d\t\n",offset,index_real+1);
					Sum_cpu.x+=h_pc[index_real].x * Sum_x - h_pc[index_real].y * Sum_y;//i;//
					Sum_cpu.y+=h_pc[index_real].x * Sum_y + h_pc[index_real].y * Sum_x;//i;//
					//printf("h_pc=%0.8f+%0.8fi\n",h_pc[index_real].x,h_pc[index_real].y);
					//printf("Sum=%0.8f+%0.8fi\n",Sum_x,Sum_y);
					//Gv_cpu[j+i*L].x+=h_pc[Strat_R+ti*L].x * Sum_x - h_pc[Strat_R+ti*L].y *Sum_y;//i;//
					//Gv_cpu[j+i*L].y+=h_pc[Strat_R+ti*L].x * Sum_y + h_pc[Strat_R+ti*L].y * Sum_x;//i;//������ô�ۼӣ������������ͷ���
				}	
			}
			Gv_cpu[j+i*L]=Sum_cpu;
		}	
	}

	QueryPerformanceCounter(&e_cpu);
	time_cpu=(double(e_cpu.QuadPart-b_cpu.QuadPart))/(double)(fp_cpu.QuadPart);
	printf("CPU-RFT����ʱ��:%f s\n\n",time_cpu);
	fd_cpu=2*Vi*(SP*M-1)/lamda;*/
	//for(int i=0; i<M;++i)printf("exp=%f+%fi\n",cos(2*PI*fd_cpu*i*Tr),sin(2*PI*fd_cpu*i*Tr));////fd
	//for(int i=0; i<SP*M;++i)printf("V=%0.8f\n",V[i]);

	//for(int i=0; i<SP*M;++i)printf("Maxoffset=%0.8f\n",0 -int( V[i]*Tr*M/delt_R));
	/*for(int i=0; i<1; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)printf("Gv_cpu[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,Gv_cpu[j+i*L].x,Gv_cpu[j+i*L].y);
	}
	////////������RFT�ı����/////////////////////////////////////////
	double *abs_Gv_cpu=(double*)malloc(sizeof(double)*SP*M*L);
	for(int i=0; i<SP*M*L; ++i)//SP*M
	{
		abs_Gv_cpu[i]=sqrt((Gv_cpu[i].x)*(Gv_cpu[i].x)+(Gv_cpu[i].y)*(Gv_cpu[i].y));
		//if(abs_Gv_cpu[i]>1e5)abs_Gv_cpu[i]=0;
	}
	FILE *fp_Gv_cpu;
	fp_Gv_cpu=fopen("d:/GV_cpu.txt","w");
	for(int i=0; i<SP*M; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)//SP*M
		{
			fprintf(fp_Gv_cpu,"%f\t",abs_Gv_cpu[j+i*L]);
		}
		fprintf(fp_Gv_cpu,"\n");
	}*/
	////////////////////////////////�����ֻ������ѹ���ʱ���Ƶ����
    hipFree(d_echo);
    hipFree(d_echo_fft);
	hipFree(d_ht);
	hipFree(d_ht_fft);
	/////////////////////////////////////////////CZT_RFT_GPU��ʼ/////////////////////////////////////////////////////
	//ϵ�������͸�ֵ////
	/*//////�μ�matlab��czt��ʵ��
	double fai=lamda*B/L/C;//��ʱ��Ƶ��ֱ���
	double *fai_a=(double*)malloc(sizeof(double)*L);//1-fai�Ľ��
	//hipfftDoubleComplex *w=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*L);//��ͬ��ʱ��Ƶ�ʵ㴦�Ļ���cztϵ��
	for(int i=0;i<L;++i)//ͬһ����ʱ�䴦��һ��fai_a
	{
		fai_a[i]=1.0f-fai*(i+1.0f);
		//w=exp(-1j*2*pi*fai_a/L);
		//w[i].x=cos(2*PI*fai_a[i]/M);
		//w[i].y=-sin(2*PI*fai_a[i]/M);
		//printf("fai_a[%d]=%0.8f\n",i+1,fai_a[i]);
	}
	double *kk=(double*)malloc(sizeof(double)*2*M-1);//�ݴ�
	double *kk2=(double*)malloc(sizeof(double)*2*M-1);//�ݴ�ƽ��
	for(int i=0;i<2*M;++i)
	{
		kk[i]=-1.0f*M+i+1;
		kk2[i]=kk[i]*kk[i]/2.0f;
		//printf("kk2[%d]=%0.8f\n",i+1,kk2[i]);
	}
	hipfftDoubleComplex *h_ww=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*L*2*M);//��ͬ��ʱ��Ƶ�ʵ㴦��cztϵ��
	hipfftDoubleComplex *h_v=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*L*2*M);//��ͬ��ʱ��Ƶ�ʵ㴦��czt 1./ϵ��
	///ww��L�У�2*M�У�����ÿһ�ж�Ӧһ����ʱ��Ƶ�㣬��ͬһ����ʱ��Ƶ�㴦��M������ز���CZT�任
	for(int i=0;i<L;++i)
	{
		//w=exp(-1j*2*pi*fai_a/L);
		//ww=w.^kk2
		//v=1./ww;
		for(int j=0; j<2*M; ++j)///���������matlba��ת�õĹ�ϵ 
		{
			//int indx_a=int(i/2/M);//ͬһ����ʱ��Ƶ��
			//int indx_kk2=i%(2*M);//��2*Mѭ��
			//printf("indx_a=%d,indx_kk2=%d\n",indx_a,indx_kk2);
			h_ww[i*2*M+j].x=cos(2*PI*fai_a[i]*kk2[j]/M);
			h_ww[i*2*M+j].y=-sin(2*PI*fai_a[i]*kk2[j]/M);
			h_v[i*2*M+j].x=cos(2*PI*fai_a[i]*kk2[j]/M);
			h_v[i*2*M+j].y=sin(2*PI*fai_a[i]*kk2[j]/M);
		}

		
	}
	
	hipfftDoubleComplex *d_ww;
	
	cudaStatus=hipMalloc((void**)&d_ww,sizeof(hipfftDoubleComplex)*2*M*L);//��GPU�˿���һ�齫�źŽ�����ά��ԭ����M*Lת�ü���ά��L*M*2������ѹƵ��ֵ��d_x
	if (cudaStatus != hipSuccess) {
        printf("d_ww hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMemcpy(d_ww,h_ww,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyHostToDevice);//�������ݵ�x����czt
	if (cudaStatus != hipSuccess) {
        printf("h_ww->d_ww hipMemcpy failed!\n");
		return 1;
    }
	/////��һ��λ�ã��Ѻ���M��wֵ��ǰ��M��wֵλ�û���//////////////
	hipfftDoubleComplex *d_ww_change;
	cudaStatus=hipMalloc((void**)&d_ww_change,sizeof(hipfftDoubleComplex)*2*M*L);//��GPU�˿���һ�齫�źŽ�����ά��ԭ����M*Lת�ü���ά��L*M*2������ѹƵ��ֵ��d_x
	if (cudaStatus != hipSuccess) {
        printf("d_ww_change hipMalloc failed!\n");
		return 1;
    }
	for(int i=0; i<L;++i)
	{
		//hipMemcpy(&d_ww_change[i*2*M],&d_ww[i*2*M+M],sizeof(hipfftDoubleComplex)*M,hipMemcpyDeviceToDevice);
		//hipMemcpy(&d_ww_change[i*2*M+M],&d_ww[i*2*M],sizeof(hipfftDoubleComplex)*M,hipMemcpyDeviceToDevice);
	}
	hipMemcpy(d_ww,d_ww_change,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyDeviceToDevice);
	cudaStatus=hipMemcpy(h_ww,d_ww,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyDeviceToHost);*/
	//////////////////////////////////////////////////////////////////
	//���ww�Ƿ���ȷ,û��2016/10/11///////////////////
	/*for(int i=0;i<1;++i)//��ʱ�䣬��
	{
		for(int j=255;j<275;++j)//��ʱ�䣬��
		{
			printf("v[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,h_v[i*2*M+j].x,h_v[i*2*M+j].y);
		}
		
	}*/

	/*hipfftDoubleComplex *d_v;
	cudaStatus=hipMalloc((void**)&d_v,sizeof(hipfftDoubleComplex)*2*M*L);//��GPU�˿���һ�齫�źŽ�����ά��ԭ����M*Lת�ü���ά��L*M*2������ѹƵ��ֵ��d_x
	if (cudaStatus != hipSuccess) {
        printf("d_v hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMemcpy(d_v,h_v,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyHostToDevice);//�������ݵ�x����czt
	if (cudaStatus != hipSuccess) {
        printf("h_v->d_v hipMemcpy failed!\n");
		return 1;
    }
	//��չx��FFTǰ��׼��
	hipfftDoubleComplex *zeros_x=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*1*M*L);//һ��M��L�е�0���󣬸�d_x���θ�ֵ
	for(int i = 0; i < 1 *M * L; ++i)
	{
		zeros_x[i].x = 0;
		zeros_x[i].y = 0;
		//if(i%100==0) printf("%0.8f+%0.8fi\n",zeros_x[i].x,zeros_x[i].y);

	}
	hipfftDoubleComplex *d_x;//pc_fft����չ����
	cudaStatus=hipMalloc((void**)&d_x,sizeof(hipfftDoubleComplex)*2*M*L);//��GPU�˿���һ�齫�źŽ�����ά��ԭ����M*Lת�ü���ά��L*M*2������ѹƵ��ֵ��d_x
	if (cudaStatus != hipSuccess) {
        printf("d_x hipMalloc failed!\n");
		return 1;
    }
	//��Ƶ����ѹ�������
	//��0����
	cudaStatus=hipMemcpy(&d_x[0],d_pc_fft,sizeof(hipfftDoubleComplex)*M*L,hipMemcpyDeviceToDevice);//�������ݵ�x����czt
	if (cudaStatus != hipSuccess) {
        printf("d_pc_fft->d_x hipMemcpy failed!\n");
		return 1;
    }//��0����x����
	cudaStatus=hipMemcpy(&d_x[M*L],zeros_x,sizeof(hipfftDoubleComplex)*1*M*L,hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        printf("zeros_x->d_x hipMemcpy failed!\n");
		return 1;
    }*/
    
	
	///////////ת��//////////////////////////////////////��ʼCZT//////////////////////////
	/*hipfftDoubleComplex *d_x_change;//ת�ú��d_x
	hipMalloc((void**)&d_x_change,sizeof(hipfftDoubleComplex)*L*2*M);//
	dim3 blcok_chang(2*M, 1);
	dim3 threadPerBlock_chang(L, 1);
	////////////��ʱ��������////////////////////
	LARGE_INTEGER b_czt_change,e_czt_change,b_czt,e_czt,b_diancheng,e_diancheng,b_fft,e_fft;//��ʱ,chang:ת��
	LARGE_INTEGER b_dianchu,e_dianchu, b_ifft, e_ifft;
	double time_czt_change,time_czt,time_diancheng,time_fft,time_ifft,time_dianchu;
	hipEvent_t stop_e_change,start_e_change;
	float time_e_change;
	hipEventCreate(&start_e_change);
	hipEventCreate(&stop_e_change);
	//QueryPerformanceCounter(&b_czt_change);
	hipEventRecord(start_e_change,0);
	ChangeVector << <blcok_chang, threadPerBlock_chang >> >(d_x, d_x_change,L);//����ת��
	hipEventRecord(stop_e_change,0);
	hipEventSynchronize(stop_e_change);
	hipEventElapsedTime(&time_e_change,start_e_change,stop_e_change);

	hipEventDestroy(stop_e_change);
	hipEventDestroy(start_e_change);
	//QueryPerformanceCounter(&e_czt_change);
	//time_czt_change =(double(e_czt_change.QuadPart-b_czt_change.QuadPart))/(double)(fp_cpu.QuadPart);*/

	///���ת���Ƿ�ɹ�////////////////////////
	/*hipfftDoubleComplex *h_x_change=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*2*M*L);//
	hipMemcpy(h_x_change,d_x_change,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyDeviceToHost);
	for(int i=0;i<1;++i)//��ʱ�䣬��
	{
		for(int j=245;j<265;++j)//��ʱ�䣬��
		{
			printf("h_x_change[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,h_x_change[i*L+j].x,h_x_change[i*L+j].y);
		}
		
	}*/
	////////////////////////////////��ʼ����FFTʵ��CZT/////////////////////////////
	//x(:,0:M-1).*ww(:,M:2*M-1)
	////���һ��//////////////////��GPU����Ҫ�����õ��������ٿռ�/////////////
	/*hipFree(d_x);
	hipfftDoubleComplex *d_y;//x.*ww���
	cudaStatus=hipMalloc((void**)&d_y,sizeof(hipfftDoubleComplex)*2*M*L);//��GPU�˿���һ�齫�źŽ�����ά��ԭ����M*Lת�ü���ά��L*M*2������ѹƵ��ֵ��d_x
	//printf("һ��ϵ����Ҫ�Ĵ洢�ռ�Ϊ%dMB\n",sizeof(hipfftDoubleComplex)*2*M*L/1024/1024);
	if (cudaStatus != hipSuccess) {
        printf("d_y hipMalloc failed!\n");
		return 1;
    }
	//cudaStatus=hipMemcpy(d_y,d_x_change,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyDeviceToDevice);//
	if (cudaStatus != hipSuccess) {
        printf("d_x->d_y hipMemcpy failed!\n");
		return 1;
    }
	dim3 blcok_xw(L,1);
	dim3 threadPerBlock_xw(2*M, 1);
	//���x.*w�Ƿ���ȷ
	hipfftDoubleComplex *h_y=(hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*2*M*L);//
	cudaStatus=hipMemcpy(h_y,d_y,sizeof(hipfftDoubleComplex)*2*M*L,hipMemcpyDeviceToHost);//
	if (cudaStatus != hipSuccess) {
        printf("d_y->h_y hipMemcpy failed!\n");
		return 1;
    }*/
	/*for(int i=0;i<1;++i)//��ʱ�䣬��
	{
		for(int j=245;j<265;++j)//��ʱ�䣬��
		{
			printf("h_y[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,h_y[i*L+j].x,h_y[i*L+j].y);
		}
	}*/

	/*hipfftDoubleComplex *d_fy,*d_fv,*d_ify,*d_sp,*d_sp_change;//x.*ww��fft���
	cudaStatus=hipMalloc((void**)&d_fy,sizeof(hipfftDoubleComplex)*2*M*L);
	if (cudaStatus != hipSuccess) {
        printf("d_fy hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_fv,sizeof(hipfftDoubleComplex)*2*M*L);
	if (cudaStatus != hipSuccess) {
        printf("d_fv hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_ify,sizeof(hipfftDoubleComplex)*2*M*L);
	if (cudaStatus != hipSuccess) {
        printf("d_ify hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_sp,sizeof(hipfftDoubleComplex)*M*L);
	if (cudaStatus != hipSuccess) {
        printf("d_sp hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_sp_change,sizeof(hipfftDoubleComplex)*M*L);
	if (cudaStatus != hipSuccess) {
        printf("d_sp_change hipMalloc failed!\n");
		return 1;
    }
	////////fft��Plan�Ͳ���׼��
	hipfftHandle Plan_fy;//��y��fft
	hipfftPlan1d(&Plan_fy,2*M,HIPFFT_Z2Z,L);
	

	hipfftHandle Plan_ify;//��y��ifft
	hipfftPlan1d(&Plan_ify,2*M,HIPFFT_Z2Z,L);
	hipfftHandle Plan_ifsp;//��sp��ifft
	hipfftPlan1d(&Plan_ifsp,L,HIPFFT_Z2Z,M);
	dim3 blcok_czt(1,L);//ÿ������һ�����뵥Ԫ
	dim3 threadPerBlock_czt(2*M,1);//ÿ�����е�һ���߳���һ������
	dim3 blcok_sp(1,M);//ÿ�����ǿ�ʱ��
	dim3 threadPerBlock_sp(L,1);//ÿ�����е�һ���߳���һ������

	

	hipEvent_t start_event_diancheng,stop_event_diancheng,start_e_czt,stop_e_czt;
	hipEvent_t start_e_fft,stop_e_fft;
	float time_e_diancheng,time_e_czt;
	float time_e_fft;
	hipEventCreate(&start_e_czt);
	hipEventCreate(&stop_e_czt);

	hipEventCreate(&start_e_fft);
	hipEventCreate(&stop_e_fft);
	//QueryPerformanceCounter(&b_czt);//czt��ʱ��ʼ
	//QueryPerformanceCounter(&b_diancheng);//��˼�ʱ��ʼ
	hipEventRecord(start_e_czt,0);
	//for (int i=0;i<1;++i)
	//{
		MulVector_xw << <blcok_xw, threadPerBlock_xw >> >(d_x,d_ww,d_y,M);//���x.*w
		//QueryPerformanceCounter(&e_diancheng);//��˼�ʱ����
		hipfftExecZ2Z(Plan_fy,d_v,d_fv,HIPFFT_FORWARD);
		//QueryPerformanceCounter(&b_fft);//fft��ʱ��ʼ
		hipEventRecord(start_e_fft,0);
		hipfftExecZ2Z(Plan_fy,d_y,d_fy,HIPFFT_FORWARD);//fft
		hipEventRecord(stop_e_fft,0);
		
		//hipEventSynchronize(stop_e_fft);
		//QueryPerformanceCounter(&e_fft);//fft��ʱ����

		MulVector <<<blcok_czt,threadPerBlock_czt>>>(d_fy, d_fv, d_fy, 2*L*M);//����ˣ���֪��Ϊʲô���к���

		//QueryPerformanceCounter(&b_ifft);//ifft��ʱ��ʼ
		hipfftExecZ2Z(Plan_fy,d_fy,d_ify,HIPFFT_BACKWARD);//ifft���껹Ҫ��Ҫ������Ϊ2*M�ĳ���
		//QueryPerformanceCounter(&e_ifft);//ifft��ʱ����

		//QueryPerformanceCounter(&b_dianchu);//�����ʱ��ʼ
		//ChuVector << <blcok_czt, threadPerBlock_czt >> >(d_ify, 2*M*L,2*M);//�����
		//QueryPerformanceCounter(&e_dianchu);//�����ʱ����

		MulVector_xw << <blcok_xw, threadPerBlock_xw >> >(d_ify,d_ww,d_sp,M);//ify(:,M:2*M-1).*w(:,M:2*M-1)

		ChangeVector<< <blcok_chang, threadPerBlock_chang >> >(d_sp,d_sp_change,M);//ת��

		hipfftExecZ2Z(Plan_ifsp,d_sp_change,d_sp_change,HIPFFT_BACKWARD);
		hipfftDoubleComplex *h_sp;//czt���ս��
		h_sp=(hipfftDoubleComplex * )malloc(sizeof(hipfftDoubleComplex)*M*L);
		cudaStatus=hipMemcpy(h_sp,d_sp_change,sizeof(hipfftDoubleComplex)*M*L,hipMemcpyDeviceToHost);//
		if (cudaStatus != hipSuccess) {
			printf("d_sp_change->h_sp hipMemcpy failed!\n");
			return 1;
		 }
		//////�ı����/////////////////////////////////
	double *h_abs_sp=(double*)malloc(sizeof(double)*M*L);
	for(int i=0; i<M*L; ++i)//SP*M
	{
		h_abs_sp[i]=sqrt((h_sp[i].x)*(h_sp[i].x)+(h_sp[i].y)*(h_sp[i].y));
		if(h_abs_sp[i]>M*Fs*Tao||h_abs_sp[i]<0) h_abs_sp[i]=0;
	}
	FILE *fp_sp;
	fp_sp=fopen("d:/sp.txt","w");
	for(int i=0; i<M; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)//SP*M
		{
			fprintf(fp_sp,"%f\t",h_abs_sp[j+i*L]);
		}
		fprintf(fp_sp,"\n");
	}
	/////////////////////////////////////////////
		//ChuVector << <blcok_sp, threadPerBlock_sp >> >(d_sp_change, M*L,L);//�����
	//}
	
	hipEventRecord(stop_e_czt,0);
	hipEventSynchronize(stop_e_czt);
	hipEventElapsedTime(&time_e_czt,start_e_czt,stop_e_czt);
	hipEventElapsedTime(&time_e_fft,start_e_fft,stop_e_fft);
	hipEventDestroy(start_e_czt);
	hipEventDestroy(stop_e_czt);
	QueryPerformanceCounter(&e_czt);//czt��ʱ����

	//time_czt = (double(e_czt.QuadPart-b_czt.QuadPart))/(double)(fp_cpu.QuadPart);
	//time_diancheng = (double(e_diancheng.QuadPart-b_diancheng.QuadPart))/(double)(fp_cpu.QuadPart);
	//time_dianchu = (double(e_dianchu.QuadPart-b_dianchu.QuadPart))/(double)(fp_cpu.QuadPart);
	//time_ifft = (double(e_ifft.QuadPart-b_ifft.QuadPart))/(double)(fp_cpu.QuadPart);
	//time_fft = (double(e_fft.QuadPart-b_fft.QuadPart))/(double)(fp_cpu.QuadPart);
	//printf("һ��ת��ʱ��=%0.8fs,CZTʱ��=%0.8fs\n",time_czt_change,time_czt);
	//printf("һ�ε��ʱ��=%0.8fs\n",time_diancheng);
	//printf("һ�ε��ʱ��=%0.8fs\n",time_dianchu);
	//printf("һ��fftʱ��=%0.8fs\n",time_fft);
	//printf("һ��ifftʱ��=%0.8fs\n",time_ifft);
	//printf("��GPU�ڹ���ʱ=%0.8fs\n",time_czt_change+time_czt);
	printf("һ��fftʱ��event��ʱ=%0.8fs\n",time_e_fft/1000);
	printf("һ��ת��ʱ��event��ʱ=%0.8fs\n",time_e_change/1000);
	printf("��GPU�ڹ���ʱevent��ʱ=%0.8fs\n",time_e_change/1000+time_e_czt/1000);
	////////////////////////////////////////////CZT_RFT_GPU����//////////////////////////////////////////////////
	/////////////////////////���CZT��ʹ�õ�ϵ��////////////////////////////////////////////////////////////////
	hipFree(d_ww);
    hipFree(d_v);
	hipFree(d_fv);
	hipFree(d_fy);
	hipFree(d_ify);
	hipFree(d_sp);
	hipFree(d_sp_change);
	hipFree(d_x_change);
	hipFree(d_y);
	free(h_echo);
	free(h_echo_fft);
	free(h_ht);
	free(h_ht_fft);
	free(h_pc);
	free(h_pc_fft);
	free(h_v);
	free(h_ww);*/
	/////////////////////////////////////////////////GPU_RFT��ʼ//////////////////////////////////////////////
	//RFT��������Ϳ��ٿռ�
	hipfftDoubleComplex *h_Gv, *d_Gv;//�����ˣ��豸�α�׼RFT�������
	h_Gv=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*SP*M*L);
	cudaStatus=hipMalloc((void**)&d_Gv,sizeof(hipfftDoubleComplex)*SP*M*L);
	if (cudaStatus != hipSuccess)
	{
		printf("d_Gv hipMalloc fail!\n");
		return 1;
	}
	double DataQ=(double)sizeof(hipfftDoubleComplex)*SP*M*L/1024/1024;//RFT���������
	printf("������%fMB\n\n",DataQ);
	dim3 block_s(M,SP);
	dim3 threadPerBlock_s(L,1);
	
	LARGE_INTEGER b1,b2,e1,e2;//��ʼʱ�䣬��������ʱ�䣬��������ʱ��
	double time1,time2;
	hipEvent_t RFT_start,RFT_end;
	float time_RFT;
	hipEventCreate(&RFT_start);
	hipEventCreate(&RFT_end);
	hipEventRecord(RFT_start,0);
	//QueryPerformanceCounter(&b1);
	RFT<<<block_s,threadPerBlock_s>>>(d_pc,d_Gv,Vi,L,SP,Tr,delt_R,lamda,d_DFT,d_offset);
	hipEventRecord(RFT_end,0);
	hipEventSynchronize(RFT_end);
	hipEventElapsedTime(&time_RFT,RFT_start,RFT_end);
	printf("GPU-RFT����ʱ��event��ʱ:%0.8f s\n",time_RFT/1000);
	hipEventDestroy(RFT_start);
	hipEventDestroy(RFT_end);
	//QueryPerformanceCounter(&e1);
	//time1=(double)(e1.QuadPart-b1.QuadPart)/(double)fp_cpu.QuadPart;
	//printf("GPU-RFT����ʱ��:%0.8f s\n",time1);

	//Ҫ���ͷŵ���������� 
	hipFree(d_pc);//
	hipFree(d_pc_fft);
	////
	//QueryPerformanceCounter(&b2);
	hipEvent_t RFT_t_start,RFT_t_end;//����
	float time_RFT_trans;//����ʱ��
	hipEventCreate(&RFT_t_start);
	hipEventCreate(&RFT_t_end);
	hipEventRecord(RFT_t_start,0);
	cudaStatus=hipMemcpy(h_Gv,d_Gv,sizeof(hipfftDoubleComplex)*SP*M*L,hipMemcpyDeviceToHost);// RFT���,���������˵��豸��
	hipEventRecord(RFT_t_end,0);
	hipEventSynchronize(RFT_t_end);
	hipEventElapsedTime(&time_RFT_trans,RFT_t_start,RFT_t_end);
	hipEventDestroy(RFT_t_start);
	hipEventDestroy(RFT_t_end);
	//QueryPerformanceCounter(&e2);
    if (cudaStatus != hipSuccess) {
        printf("d_Gv->h_Gv hipMemcpy failed!\n Error Code:%d",cudaStatus);
		return 1;
    }
	printf("RFT���ݴ���ʱ��event��ʱ:%0.8f s\n",time_RFT_trans/1000);
	printf("RFT����ʱevent��ʱ:%0.8f s\n",(time_RFT_trans+time_RFT)/1000);
	//time2=(double)(e2.QuadPart-b2.QuadPart)/(double)fp_cpu.QuadPart;
	//printf("���ݴ���ʱ��:%0.8f s\n",time2);
	//printf("����ʱ:%0.8f s\n",(time2+time1));
	double TransSpeed =DataQ/time_RFT_trans;
	//int TransSpeed =DataQ/time2;
	printf("�����ٶ�Ϊ%.2fMB s\n\n",TransSpeed*1000);

	//double Speedup=time_cpu/(time_RFT_trans+time_RFT)*1000;
	//printf("���ٱ�:%f\n",Speedup);
	/*for(int i=10*M-2; i<10*M-1; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)printf("h_Gv[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,h_Gv[j+i*L].x,h_Gv[j+i*L].y);
	}*/
	/////����Դ�///////////////////////////////////////////////////////////
	//hipFree(d_Gv);
	/////////////////////////////////////////////////////////////////////////////�ı����/////
	float *h_abs_Gv=(float*)malloc(sizeof(float)*SP*M*L);
	for(int i=0; i<SP*M*L; ++i)//SP*M
	{
		h_abs_Gv[i]=sqrt((h_Gv[i].x)*(h_Gv[i].x)+(h_Gv[i].y)*(h_Gv[i].y));
		if(h_abs_Gv[i]<0) h_abs_Gv[i]=0;//h_abs_Gv[i]>M*Fs*Tao||
	}
	FILE *fp_Gv;
	fp_Gv=fopen("d:/GV.txt","w");
	for(int i=0; i<SP*M; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)//SP*M
		{
			fprintf(fp_Gv,"%.2f\t",h_abs_Gv[j+i*L]);
		}
		fprintf(fp_Gv,"\n");
	}
	//free(Gv_cpu);
	//free(h_abs_Gv);
	//free(h_Gv);
    return 0;
}

