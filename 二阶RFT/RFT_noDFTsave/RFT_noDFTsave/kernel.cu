#include "hip/hip_runtime.h"
//�������ȴ洢��DFTϵ����ͨ��ע�����ı�����DFT�洢��ʽ
//
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <Windows.h>
#define PI 3.14159265358979323846
#define M 256//���������
///kernel ����
//���
__global__ void MulVector(hipfftComplex *a, hipfftComplex *b, hipfftComplex *c,int size)
{
	int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����
	int id_tx = threadIdx.x;//���ڿ���̺߳ź��ᣬ�ڼ������뵥Ԫ
	int id_ty = threadIdx.y;//���ڿ���̺߳����ᣬ���ڵڼ�������
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤�ȣ���L
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤�ȣ���N

	int index_real = (id_tx+threadPerBlock_x*id_ty)+id_by*threadPerBlock_x*threadPerBlock_y; //����������ֵ=�ڿ���+����Ŀ���
	int index_ht = id_tx;//���ݺ���������
	int index_echo = index_real;// �ز�����������
	if (index_real < size)
	{
		c[index_real].x = a[index_ht].x * b[index_echo].x - a[index_ht].y * b[index_echo].y;//blockDim.x;//index_ht;//
		c[index_real].y = a[index_ht].x * b[index_echo].y + a[index_ht].y * b[index_echo].x;//blockDim.y;//index_real;//
	}
}
//���
__global__ void ChuVector(hipfftComplex *a, int size,int L)
{
    int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����
	int id_tx = threadIdx.x;//���ڿ���̺߳ź��ᣬ�ڼ������뵥Ԫ
	int id_ty = threadIdx.y;//���ڿ���̺߳����ᣬ���ڵڼ�������
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤�ȣ���L
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤�ȣ���N
	int index_real = (id_tx+threadPerBlock_x*id_ty)+id_by*threadPerBlock_x*threadPerBlock_y; //����������ֵ=�ڿ���+����Ŀ���
	if (index_real < size)
	{
		a[index_real].x = a[index_real].x / L;
		a[index_real].y = a[index_real].y / L;
	}
}
//��׼RFT///////////////////////////////////////////////////////
//                        ����ز���              RFT�����     �����ٶȲ������ٶ������ľ��뵥Ԫ���ֵ,�����ظ����,���뵥Ԫ��С������
__global__ void RFT(hipfftComplex *pc, hipfftComplex *Gv, float Vi, int L, int SP, float Tr , float delt_R, float lamda,hipfftComplex *d_DFT, float *d_offset, float V_offset)
{
	hipfftComplex Sum={0,0};
	int id_bx = blockIdx.x;//���ڵĿ�ź���,�ڼ�����
	int id_by = blockIdx.y;//���ڵĿ������,�ڼ�����
	int id_tx = threadIdx.x;//���ڿ���̺߳ź���,��ʼ���뵥Ԫ
	int id_ty = threadIdx.y;//���ڿ���̺߳�����,���ڵڼ����ٶ�
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤,��ʼ����
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤��,һ��������ѵ��ٶȸ���
	int BlockPerGid_x= gridDim.x;//grid�еĿ��������
	int BlockPerGid_y= gridDim.y;//grid�еĿ��������
	//__shared__ hipfftComplex Pc_share[M];
	//����������ֵ=      �ڿ���                      +           ����Ŀ���        *           ��Ĵ�С
	int index_real =(id_tx + id_ty*threadPerBlock_x) + (id_bx + id_by*BlockPerGid_x) * (threadPerBlock_x*threadPerBlock_y);
	int V_index = id_ty + threadPerBlock_y * (id_bx + id_by*BlockPerGid_x) ;//�ٶ�����
	float V=Vi*V_index-V_offset;//���߳��������ٶ�
	int Strat_R=id_tx;//��ʼ���뵥Ԫ
	int maxoffset=floor(V*M*Tr/delt_R+0.5);//�������߶�
	float Sum_x=0;
	float Sum_y=0;
	int   offset=0;
	if (Strat_R-maxoffset>=0 && index_real<SP*M*L && Strat_R<L)//û���������������ͼ���
	{
		float fd = 2*V/lamda;//û��DFTϵ��
		for(int i=0;i<M;++i)
		{
			offset=floor(V*i*Tr/delt_R+0.5);
			Strat_R=Strat_R-d_offset[i+id_bx*M+id_by*M*M];
			
			Sum_x=cos(2*PI*fd*i*Tr);//DFTϵ�����߳������
			Sum_y=sin(2*PI*fd*i*Tr);//DFTϵ�����߳������
			Sum.x+=pc[Strat_R+i*L].x * Sum_x - pc[Strat_R+i*L].y * Sum_y;//i;//
			Sum.y+=pc[Strat_R+i*L].x * Sum_y + pc[Strat_R+i*L].y * Sum_x;//i;//
			//�������ȴ洢��DFTϵ����ͨ��ע�����ı�����DFT�洢��ʽ
			//Sum.x+=pc[Strat_R+i*L].x * d_DFT[i+id_bx*M+id_by*M*M].x - pc[Strat_R+i*L].y * d_DFT[i+id_bx*M+id_by*M*M].y;//i;//
			//Sum.y+=pc[Strat_R+i*L].x * d_DFT[i+id_bx*M+id_by*M*M].y + pc[Strat_R+i*L].y * d_DFT[i+id_bx*M+id_by*M*M].x;//i;//
			//Strat_R=id_tx;
		}
		Gv[index_real]=Sum;
		Sum.x= 0;
		Sum.y= 0;
	}
	else 
	{
		Gv[index_real].x=0;
		Gv[index_real].y=0;
	}
	Sum.x= 0;
	Sum.y= 0;
}
//ת��
__global__ void ChangeVector(hipfftComplex *a,hipfftComplex *b,int L)//����ת��
{
	__shared__ hipfftComplex share_block[600];//�潻���õ�
	//hipfftComplex temp;//���õ���ʱ���������ڼĴ�������
	int id_tx = threadIdx.x;//���ڿ���̺߳ź���
	int id_bx = blockIdx.x;//���ڵĿ������
	int threadPerBlock_x = blockDim.x;//���ڳߴ�ĺ��᳤��
	if(id_tx < L && id_bx < 2 * M)
	{
		int index_out = id_tx + id_bx*threadPerBlock_x; //���������
		share_block[id_tx] = a[index_out];
	}

	__syncthreads();//���Ϊ������߳�ͬ��

	id_tx = threadIdx.x;//���ڿ���̺߳ź���
	id_bx = blockIdx.x;//���ڵĿ������
	int threadPerBlock_y = blockDim.y;//���ڳߴ�����᳤��
	if(id_tx < L && id_bx < 2 * M)
	{
		int index_in = id_tx*(2*M)+id_bx;//����������
		b[index_in] = share_block[id_tx];
	}

	//temp = a[index_out];
	//a[index_out] = a[index_in];
	//a[index_in] = temp;

}
int main()
{
	//��ʱ��������
	LARGE_INTEGER fp_cpu;//cpu��Ƶ
	QueryPerformanceFrequency(&fp_cpu);//��ȡ��Ƶ
		///GPU��Ϣ˵��
	int MTPB = 1024;//��block���֧���߳���
	int MaxThreadBlockSize[3]={1024,1024,64};//���Ŀ����֧꣬��1024*1024*64����
		/*int dev=1;
	hipDeviceProp_t prop;
	hipGetDevice(&dev);
	printf("GPU�ͺ�:%s\n",prop.name);
	printf("��block���֧���߳���:%d\n",prop.maxThreadsPerBlock);*/
	///���������͸�ֵ
	float fc=100e6;//��Ƶ
	float B=4e6;//����
	float Tao=128e-6;//����
	float Fs=1*B;//����Ƶ��
	float Ts=1/Fs;
	//printf("Ts=%0.8f\n",Ts);
	float mu=B/Tao;//��Ƶ��
	float C=3e8;//����
	float delt_R=C/(2*Fs);//����ֱ���
	float R_start=359*delt_R;//��ʼ����
	float lamda=C/fc;//����
	float PRF=500;//�����ظ�����
	float Tr=1/PRF;//�����ظ�ʱ��
	float Vr=2000;//��ʼ�ٶ�
	printf("Vr=%f\n",Vr);
	float a=0;//���ٶ�
	int    L=Tao*Fs;//���뵥Ԫ���������������
	printf("L=%d\n",L);
	int    N=MTPB/L;//�����߳�����ά����
	printf("N=%d\n",N);
	float *t=(float*)malloc(sizeof(float)*L);//��ʱ��
	float *delt_t=(float*)malloc(sizeof(float)*M);//�ӳ�ʱ��


	for(int i=0;i<L;++i)
	{
		t[i]=-Tao/2+Ts*i;	
		//t[i]=Ts*i;	
	   // printf("t[%d]=%0.8f\n",i+1,t[i]);
	}
	
	///��������/////////////////////////////////////////////
	
	hipfftComplex  *h_echo,*d_echo; //������,�豸�λز�ʱ��
	hipfftComplex  *h_echo_fft,*d_echo_fft; //������,�豸�λز�Ƶ��

	hipfftComplex  *h_ht,*d_ht;    //������,�豸����ѹʱ��ϵ��
	hipfftComplex  *h_ht_fft,*d_ht_fft;    //������,�豸����ѹʱ��ϵ��

	hipfftComplex  *h_pc,*d_pc;   //������,�豸��ѹʱ����
	hipfftComplex  *h_pc_fft,*d_pc_fft;//������,�豸��ѹƵ����

	//�����ڴ棬�Դ�////////////////////////
	hipError_t cudaStatus;//״̬��¼
    h_echo=(hipfftComplex *)malloc(sizeof(hipfftComplex )*M*L);//�����˻ز�ʱ��
	h_echo_fft=(hipfftComplex *)malloc(sizeof(hipfftComplex )*M*L);//�����˻ز�Ƶ��

	h_ht=(hipfftComplex *)malloc(sizeof(hipfftComplex )*L);//��������ѹʱ��ϵ��
	h_ht_fft=(hipfftComplex *)malloc(sizeof(hipfftComplex )*L);//��������ѹƵ��ϵ��

	h_pc=(hipfftComplex *)malloc(sizeof(hipfftComplex )*M*L);//��������ѹʱ����
	h_pc_fft=(hipfftComplex *)malloc(sizeof(hipfftComplex )*M*L);//��������ѹƵ����

    cudaStatus=hipMalloc((void**)&d_echo,sizeof(hipfftComplex )*M*L);//�豸�λز�ʱ�򿪱��Դ�	    
    if (cudaStatus != hipSuccess) {
        printf( "d_echo hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_echo_fft,sizeof(hipfftComplex )*M*L);//�豸�λز�Ƶ�򿪱��Դ�
	if (cudaStatus != hipSuccess) {
        printf( "d_echo_fft hipMalloc failed!\n");
		return 1;
    }

	cudaStatus=hipMalloc((void**)&d_pc_fft,sizeof(hipfftComplex )*M*L);//�豸�λز���ѹƵ��
	if (cudaStatus != hipSuccess) {
        printf( "d_pc_fft hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_pc,sizeof(hipfftComplex )*M*L);//�豸�λز���ѹʱ��
	if (cudaStatus != hipSuccess) {
        printf( "d_pc hipMalloc failed!\n");
		return 1;
    }

	cudaStatus=hipMalloc((void**)&d_ht,sizeof(hipfftComplex )*L);//�豸�λز���ѹʱ��ϵ���Դ�
	
	if (cudaStatus != hipSuccess) {
        printf( "d_ht hipMalloc failed!\n");
		return 1;
    }
	cudaStatus=hipMalloc((void**)&d_ht_fft,sizeof(hipfftComplex )*L);//�豸�λز���ѹƵ��ϵ���Դ�
	if (cudaStatus != hipSuccess) {
        printf( "d_ht_fft hipMalloc failed!\n");
		return 1;
    }
	
	///��ʼ��ֵ///////
	///��������ѹʱ��ϵ��
	for(int i=0;i<=L-1;++i)///
	{
		h_ht[i].x=cos(2*PI*(mu/2)*t[i]*t[i]);
		h_ht[i].y=sin(2*PI*(mu/2)*t[i]*t[i]);
	}
	//���ݺ�����ת����
	/*hipfftComplex t_h;//�����õ���ʱ����
	for (int i = 0; i < L / 2; ++i)//���ݺ���
	{
		if (L - i - 1 >= 0)
		{
			t_h.x = h_ht[i].x;
			h_ht[i].x = h_ht[L - i - 1].x;
			h_ht[L - i - 1].x = t_h.x;
			t_h.y = -1 * h_ht[i].y;//����Ӹ���
			h_ht[i].y = -1 * h_ht[L - i - 1].y;
			h_ht[L - i - 1].y = t_h.y;
		}
	}*/
   //for(int i=0;i<=L-1;++i)printf("h_ht[%d]=%0.8f+%0.8fi\n",i+1,h_ht[i].x,h_ht[i].y);
	///�ز���ֵ
	for(int j=0;j<M;++j)//�ӳټ���
	{
			Vr=Vr+a*Tr*j;
		    delt_t[j]=2*(R_start+Vr*Tr*j)/C;
	}
	for(int i=0;i<M;++i)//��ʱ��
	{
		for(int j=0;j<L; ++j)//��ʱ��
		{
			h_echo[j+i*L].x=cos(2*PI*(mu/2*(t[j]+delt_t[i])*(t[j]+delt_t[i])+fc*delt_t[i]));
			h_echo[j+i*L].y=-sin(2*PI*(mu/2*(t[j]+delt_t[i])*(t[j]+delt_t[i])+fc*delt_t[i]));
			//printf("h_echo[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_echo[j+i*L].x,h_echo[j+i*L].y);
			//if((j+1)%3==0) printf("\n");
		}
		//printf("\n");
	}
	//for(int i=L;i<=2*L-1;++i)printf("h_echo[%d]=%0.8f+%0.8fi\n",i+1,h_echo[i].x,h_echo[i].y);
	//���������ݿ������豸��
	cudaStatus=hipMemcpy(d_echo,h_echo,sizeof(hipfftComplex )*M*L,hipMemcpyHostToDevice);//�ز����������˵��豸��
	if (cudaStatus != hipSuccess) {
        printf("h_echo->d_echo hipMemcpy failed!\n");
		return 1;
    }
	cudaStatus=hipMemcpy(d_ht,h_ht,sizeof(hipfftComplex )*L,hipMemcpyHostToDevice);// ��ѹʱ��ϵ��,���������˵��豸��
    if (cudaStatus != hipSuccess) {
        printf("h_ht->d_ht hipMemcpy failed!\n");
		return 1;
    }
	//�Իز�����ѹϵ����fft�任��Ƶ����Ƶ�������ѹ
	//����fft�ƻ�
	LARGE_INTEGER b_pc1,e_pc1,b_pc2,e_pc2;//��ѹ��ʱ
	float time_pc1,time_pc2;
	hipfftHandle plan_ML,plan_L;//��L��M����fft����L��1����fft�ƻ�
	hipfftResult Result_fft_ML,Result_fft_L;//���ִ�н��������ֵ���ͻ�����hipError_t�����˹���
	hipfftPlan1d(&plan_ML,L,HIPFFT_C2C,M);//���ûز�fft�ƻ�
	QueryPerformanceCounter(&b_pc1);
	Result_fft_ML=hipfftExecC2C(plan_ML,d_echo,d_echo_fft,HIPFFT_FORWARD);//�Իز�����FFT
	//��FFT��0��fsǧ����matlab��fftshift������ݱȶ�
	/*if (Result_fft_ML != hipSuccess)
	{
		printf("�ز�fft����!\n");	
		return 1;
	}*/
	/*cudaStatus=hipMemcpy(h_echo_fft,d_echo,sizeof(hipfftComplex )*M*L,hipMemcpyDeviceToHost);//�ز�Ƶ�������豸�ε�������
	if (cudaStatus != hipSuccess) {
        printf("d_echo_fft->h_echo_fft hipMemcpy failed!\n");
		return 1;
    }*/
	///
	/*for(int i=0;i<M;++i)//��ʱ��
	{
		for(int j=0;j<L; ++j)//��ʱ��
		{
			printf("h_echo_fft[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_echo_fft[j+i*L].x,h_echo_fft[j+i*L].y);
			if((j+1)%2==0) printf("\n");
		}
		printf("\n");
	}*/
	//for(int i=0;i<=L-1;++i)printf("h_echo_fft[%d]=%0.8f+%0.8fi\n",i+1,h_echo_fft[i].x,h_echo_fft[i].y);
	hipfftPlan1d(&plan_L,L,HIPFFT_C2C,1);
	Result_fft_L=hipfftExecC2C(plan_L,d_ht,d_ht_fft,HIPFFT_FORWARD);//����ѹϵ������FFT HIPFFT_BACKWARD HIPFFT_FORWARD
	if (Result_fft_L != hipSuccess)
	{
		printf("��ѹϵ��fft����!\n");	
		return 1;
	}
	/*cudaStatus=hipMemcpy(h_ht_fft,d_ht_fft,sizeof(hipfftComplex )*L,hipMemcpyDeviceToHost);//��ѹϵ��Ƶ�������豸�ε�������
	if (cudaStatus != hipSuccess) {
        printf("d_ht_fft->h_ht_fft hipMemcpy failed!\n");
		return 1;
    }
	for(int j=0;j<L; ++j)
	{
			printf("h_ht_fft[%d]=%0.8f+%0.8fi \n ",j+1,h_ht_fft[j].x,h_ht_fft[j].y);
	}*/
	//������ѹ/////////////////////////////////////////////
	//GPU���
	dim3 blcok(1,M);//ÿ������һ�����뵥Ԫ
	dim3 threadPerBlock(L,1);//ÿ�����е�һ���߳���һ������
	MulVector <<<blcok,threadPerBlock>>>(d_ht_fft, d_echo_fft, d_pc_fft, L*M);//����ˣ���֪��Ϊʲô���к���
	//������Ƿ���ȷ/////////////////////
	/*cudaStatus=hipMemcpy(h_pc_fft,d_pc_fft,sizeof(hipfftComplex )*M*L,hipMemcpyDeviceToHost);//��ѹƵ�������豸�ε�������
    if (cudaStatus != hipSuccess) {
        printf("d_pc_fft->h_pc_fft hipMemcpy failed!\n");
		return 1;
    }*/
	/*for(int i=0;i<M;++i)
	{
		for(int j=0;j<L; ++j)
		{
			printf("h_pc_fft[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_pc_fft[j+i*L].x,h_pc_fft[j+i*L].y);
			printf("\n");
		}
		printf("\n");
	}*/

	Result_fft_ML=hipfftExecC2C(plan_ML,d_pc_fft,d_pc,HIPFFT_BACKWARD);//��ѹIFFT
	if (Result_fft_ML != hipSuccess)
	{
		printf("�ز�fft����!\n");	
		return 1;
	}
	ChuVector << <blcok, threadPerBlock >> >(d_pc, M*L,L);//�����
	QueryPerformanceCounter(&e_pc1);
	time_pc1=(float)(e_pc1.QuadPart-b_pc1.QuadPart)/(float)fp_cpu.QuadPart;
	QueryPerformanceCounter(&b_pc2);
	cudaStatus=hipMemcpy(h_pc,d_pc,sizeof(hipfftComplex )*M*L,hipMemcpyDeviceToHost);//��ѹƵ�������豸�ε�������
	QueryPerformanceCounter(&e_pc2);
	time_pc2=(float)(e_pc2.QuadPart-b_pc2.QuadPart)/(float)fp_cpu.QuadPart;
	printf("��ѹ��ʱ:%f s\n",time_pc1);
	printf("���ݴ�����ʱ:%f s\n",time_pc2);
	printf("����ʱ:%f s\n\n",time_pc1+time_pc2);
    /*if (cudaStatus != hipSuccess) {
        printf("d_pc_fft->h_pc_fft hipMemcpy failed!\n");
		return 1;
    }
	for(int i=33;i<34;++i)
	{
		for(int j=0;j<L; ++j)
		{
			printf("h_pc[%d][%d]=%0.8f+%0.8fi  ",i+1,j+1,h_pc[j+i*L].x,h_pc[j+i*L].y);
			printf("\n");
		}
		printf("\n");
	}*/
	/////////��ѹ�������////////////////////////////////////
	float *h_abs_pc=(float*)malloc(sizeof(float)*M*L);
	for(int i=0; i<M*L; ++i)h_abs_pc[i]=sqrt(h_pc[i].x*h_pc[i].x+h_pc[i].y*h_pc[i].y);
	FILE *fp_pc;
	fp_pc=fopen("d:/Pc.txt","w");
	for(int i=0; i<M; ++i)
	{
		for(int j=0; j<L; ++j)
		{
			fprintf(fp_pc,"%0.8f\t",h_abs_pc[j+i*L]);
		}
		fprintf(fp_pc,"\n");
	}
	//////////////////////////////////////////��ѹ���////////////////////////////////////////////////
	///////////////////////////////////////////RFT///////////////////////////////////////////////////////
	//������������
	float Vb = lamda/2/Tr;//ä��
	//printf("Vb=%f\n",Vb);
	float Vi = Vb/M; //�ٶ���������,�ɵ�
	//printf("Vi=%f\n",Vi);
	int    SP =400;//1024*6/M;   //������ä���������//
	float V_offset =SP/2*Vb;
	printf("SP=%d\n",SP);
	printf("�ٶ�����[%f,%f]\n",-SP/2*Vb,SP/2*Vb);

	/////����SP�������ٶȡ���Ӧ�Ķ����մ洢�ã��ڼ����Ӧ��DFTϵ���ٴ洢��//////////////
	float *h_fd, *h_offset, *d_offset;
	h_fd=(float *)malloc(sizeof(float )*M*SP);//������fd�洢
	h_offset=(float *)malloc(sizeof(float )*M*SP*M);//������û���ٶ���,ÿ�������ظ����ڵ�ƫ������
	cudaStatus=hipMalloc((void**)&d_offset,sizeof(hipComplex )*M*SP*M);//�豸��û���ٶ���,ÿ�������ظ����ڵ�ƫ������
	if (cudaStatus != hipSuccess) {
        printf("d_offset hipMalloc failed!\n");
		return 1;
    }
	////����fd,ƫ����//////////
	for(int i=0;i<M*SP;++i)
	{
		h_fd[i] = 2*Vi*i/lamda;
		for(int j=0; j<M; ++j)
		{
			h_offset[i*M+j]=floor(Vi*i*j*Tr/delt_R+0.5);
		}
		
	}
	cudaStatus=hipMemcpy(d_offset,h_offset,sizeof(float)*M*SP*M,hipMemcpyHostToDevice);//��������������ƫ�������豸��
		if (cudaStatus != hipSuccess) {
        printf("h_offset->d_offset  hipMemcpy failed!\n");
		return 1;
    }
	///////////////����DFTϵ��///////////////
	hipComplex *h_DFT,*d_DFT;
	h_DFT=(hipComplex *)malloc(sizeof(hipComplex)*M*SP*M);//������DFT�洢
	cudaStatus=hipMalloc((void**)&d_DFT,sizeof(hipComplex )*M*SP*M);//�豸��DFT�洢
	for(int i=0; i<M*SP; ++i)
	{
		for(int j=0; j<M; ++j)
		{
			h_DFT[i*M+j].x=cos(2*PI*h_fd[i]*j*Tr);
			h_DFT[i*M+j].y=sin(2*PI*h_fd[i]*j*Tr);
		}	
	}
	cudaStatus=hipMemcpy(d_DFT,h_DFT,sizeof(hipfftComplex)*M*SP*M,hipMemcpyHostToDevice);//��������������DFTϵ�����豸��
	if (cudaStatus != hipSuccess) {
        printf("h_DFT->d_DFT  hipMemcpy failed!\n");
		return 1;
    }
	free(h_DFT);
	free(h_fd);

	////////////////////////////////////////CPU_RFT//////////////////////////////////////////////////////////
	/*int    Strat_R=0;
	float Sum_x,Sum_y;
	long float offset;
	float2 Sum_cpu;
	hipfftComplex *Gv_cpu=(hipfftComplex*)malloc(sizeof(hipfftComplex)*SP*M*L);//V*L,�ٶ�-���뵥Ԫ��ά
	LARGE_INTEGER b_cpu,e_cpu;//��ʱ
	float time_cpu;
	float fd_cpu;
	float *V=(float*)malloc(sizeof(float)*SP*M);
	int    index_real;
	QueryPerformanceCounter(&b_cpu);
	for(int i=0; i<SP*M; ++i)//�ٶ�
	{
		V[i]=Vi*i-V_offset;
		fd_cpu=2*V[i]/lamda;//������
		//printf("V=%f\t",V[i]);
		//printf("fd_cpu=%f\t",fd_cpu);
		for(int j=0; j<L; ++j)//���뵥Ԫ
		{
			Sum_cpu.x=0;
			Sum_cpu.y=0;
			Strat_R=j;
			//printf("Strat_R=%d\n",Strat_R);
			if(Strat_R -floor(V[i]*Tr*M/delt_R+0.5)>=10 && Strat_R + V[i]*Tr*M/delt_R<L-10)
			{
				for(int ti=0; ti<M; ++ti)
				{
					offset=floor(V[i]*ti*Tr/delt_R+0.5);
					Strat_R=Strat_R-floor(offset);
					Sum_x=cos(2*PI*fd_cpu*ti*Tr);
					Sum_y=sin(2*PI*fd_cpu*ti*Tr);
					index_real=Strat_R+ti*L;
					Strat_R=j;
					//printf("offset=%d\t index_real=%d\t\n",offset,index_real+1);
					Sum_cpu.x+=h_pc[index_real].x * Sum_x - h_pc[index_real].y * Sum_y;//i;//
					Sum_cpu.y+=h_pc[index_real].x * Sum_y + h_pc[index_real].y * Sum_x;//i;//
					//printf("h_pc=%0.8f+%0.8fi\n",h_pc[index_real].x,h_pc[index_real].y);
					//printf("Sum=%0.8f+%0.8fi\n",Sum_x,Sum_y);
					//Gv_cpu[j+i*L].x+=h_pc[Strat_R+ti*L].x * Sum_x - h_pc[Strat_R+ti*L].y *Sum_y;//i;//
					//Gv_cpu[j+i*L].y+=h_pc[Strat_R+ti*L].x * Sum_y + h_pc[Strat_R+ti*L].y * Sum_x;//i;//������ô�ۼӣ������������ͷ���
				}	
			}
			Gv_cpu[j+i*L]=Sum_cpu;
		}	
	}

	QueryPerformanceCounter(&e_cpu);
	time_cpu=(float(e_cpu.QuadPart-b_cpu.QuadPart))/(float)(fp_cpu.QuadPart);
	printf("CPU-RFT����ʱ��:%f s\n\n",time_cpu);
	fd_cpu=2*Vi*(SP*M-1)/lamda;*/
	//for(int i=0; i<M;++i)printf("exp=%f+%fi\n",cos(2*PI*fd_cpu*i*Tr),sin(2*PI*fd_cpu*i*Tr));////fd
	//for(int i=0; i<SP*M;++i)printf("V=%0.8f\n",V[i]);

	//for(int i=0; i<SP*M;++i)printf("Maxoffset=%0.8f\n",0 -int( V[i]*Tr*M/delt_R));
	/*for(int i=0; i<1; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)printf("Gv_cpu[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,Gv_cpu[j+i*L].x,Gv_cpu[j+i*L].y);
	}*/
	////////������RFT�ı����/////////////////////////////////////////
	/*float *abs_Gv_cpu=(float*)malloc(sizeof(float)*SP*M*L);
	for(int i=0; i<SP*M*L; ++i)//SP*M
	{
		abs_Gv_cpu[i]=sqrt((Gv_cpu[i].x)*(Gv_cpu[i].x)+(Gv_cpu[i].y)*(Gv_cpu[i].y));
		if(abs_Gv_cpu[i]>1e6)abs_Gv_cpu[i]=0;
	}
	FILE *fp_Gv_cpu;
	fp_Gv_cpu=fopen("d:/GV_cpu.txt","w");
	for(int i=0; i<SP*M; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)//SP*M
		{
			fprintf(fp_Gv_cpu,"%f\t",abs_Gv_cpu[j+i*L]);
		}
		fprintf(fp_Gv_cpu,"\n");
	}*/
	////////////////////////////////�����ֻ������ѹ���ʱ���Ƶ����
    hipFree(d_echo);
    hipFree(d_echo_fft);
	hipFree(d_ht);
	hipFree(d_ht_fft);
	/////////////////////////////////////////////////GPU_RFT��ʼ//////////////////////////////////////////////
	//RFT��������Ϳ��ٿռ�
	hipfftComplex *h_Gv, *d_Gv;//�����ˣ��豸�α�׼RFT�������
	h_Gv=(hipfftComplex*)malloc(sizeof(hipfftComplex)*SP*M*L);
	cudaStatus=hipMalloc((void**)&d_Gv,sizeof(hipfftComplex)*SP*M*L);
	if (cudaStatus != hipSuccess)
	{
		printf("d_Gv hipMalloc fail!\n");
		return 1;
	}
	float DataQ=(float)sizeof(hipfftComplex)*SP*M*L/1024/1024;//RFT���������
	printf("������%fMB\n\n",DataQ);
	dim3 block_s(M,SP);//M,SP
	dim3 threadPerBlock_s(L,1);
	
	LARGE_INTEGER b1,b2,e1,e2;//��ʼʱ�䣬��������ʱ�䣬��������ʱ��
	float time1,time2;
	hipEvent_t RFT_start,RFT_end;
	float time_RFT;
	hipEventCreate(&RFT_start);
	hipEventCreate(&RFT_end);
	hipEventRecord(RFT_start,0);
	//QueryPerformanceCounter(&b1);
	RFT<<<block_s,threadPerBlock_s>>>(d_pc,d_Gv,Vi,L,SP,Tr,delt_R,lamda,d_DFT,d_offset,V_offset);
	hipEventRecord(RFT_end,0);
	hipEventSynchronize(RFT_end);
	hipEventElapsedTime(&time_RFT,RFT_start,RFT_end);
	printf("GPU-RFT����ʱ��event��ʱ:%0.8f s\n",time_RFT/1000);
	hipEventDestroy(RFT_start);
	hipEventDestroy(RFT_end);
	//QueryPerformanceCounter(&e1);
	//time1=(float)(e1.QuadPart-b1.QuadPart)/(float)fp_cpu.QuadPart;
	//printf("GPU-RFT����ʱ��:%0.8f s\n",time1);

	//Ҫ���ͷŵ���������� 
	hipFree(d_pc);//
	hipFree(d_pc_fft);
	////
	//QueryPerformanceCounter(&b2);
	hipEvent_t RFT_t_start,RFT_t_end;//����
	float time_RFT_trans;//����ʱ��
	hipEventCreate(&RFT_t_start);
	hipEventCreate(&RFT_t_end);
	hipEventRecord(RFT_t_start,0);
	cudaStatus=hipMemcpy(h_Gv,d_Gv,sizeof(hipfftComplex)*SP*M*L,hipMemcpyDeviceToHost);// RFT���,���������˵��豸��
	hipEventRecord(RFT_t_end,0);
	hipEventSynchronize(RFT_t_end);
	hipEventElapsedTime(&time_RFT_trans,RFT_t_start,RFT_t_end);
	hipEventDestroy(RFT_t_start);
	hipEventDestroy(RFT_t_end);
	//QueryPerformanceCounter(&e2);
    if (cudaStatus != hipSuccess) {
        printf("d_Gv->h_Gv hipMemcpy failed!\n Error Code:%d",cudaStatus);
		return 1;
    }
	printf("RFT���ݴ���ʱ��event��ʱ:%0.8f s\n",time_RFT_trans/1000);
	printf("RFT����ʱevent��ʱ:%0.8f s\n",(time_RFT_trans+time_RFT)/1000);
	//time2=(float)(e2.QuadPart-b2.QuadPart)/(float)fp_cpu.QuadPart;
	//printf("���ݴ���ʱ��:%0.8f s\n",time2);
	//printf("����ʱ:%0.8f s\n",(time2+time1));
	float TransSpeed =DataQ/time_RFT_trans;
	//int TransSpeed =DataQ/time2;
	printf("�����ٶ�Ϊ%.2fMB s\n\n",TransSpeed*1000);

//	float Speedup=time_cpu/(time_RFT_trans+time_RFT)*1000;
	//printf("���ٱ�:%f\n",Speedup);
	/*for(int i=10*M-2; i<10*M-1; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)printf("h_Gv[%d][%d]=%0.8f+%0.8fi\n",i+1,j+1,h_Gv[j+i*L].x,h_Gv[j+i*L].y);
	}*/
	/////����Դ�///////////////////////////////////////////////////////////
	//hipFree(d_Gv);
	/////////////////////////////////////////////////////////////////////////////�ı����/////
	float *h_abs_Gv=(float*)malloc(sizeof(float)*SP*M*L);
	for(int i=0; i<SP*M*L; ++i)//SP*M
	{
		h_abs_Gv[i]=sqrt((h_Gv[i].x)*(h_Gv[i].x)+(h_Gv[i].y)*(h_Gv[i].y));
		if(h_abs_Gv[i]<0) h_abs_Gv[i]=0;//h_abs_Gv[i]>M*Fs*Tao||
	}
	FILE *fp_Gv;
	fp_Gv=fopen("d:/GV.txt","w");
	for(int i=0; i<SP*M; ++i)//SP*M
	{
		for(int j=0; j<L; ++j)//SP*M
		{
			fprintf(fp_Gv,"%.2f\t",h_abs_Gv[j+i*L]);
		}
		fprintf(fp_Gv,"\n");
	}
	//free(Gv_cpu);
	//free(h_abs_Gv);
	//free(h_Gv);
    return 0;
}

