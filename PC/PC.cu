#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include "PC.h"
#include <mex.h>
#pragma comment(lib,"cufft.lib")
__global__ void DianCheng(hipfftDoubleComplex *a,  hipfftDoubleComplex *b, hipfftDoubleComplex *c,int M, int L)//��˵�GPU����
{
      int tx = threadIdx.y;
      int by = blockIdx.x;
      int i=by*L+tx;
      if (i<=M*L)
    {
       c[i].x = a[tx].x * b[i].x-a[tx].y*b[i].y;
       c[i].y = a[tx].x * b[i].y+a[tx].y*b[i].x;
    }   
}
void pc(double *realh, double *imagh, double *realecho, double *imagecho, 
        double *realpc, double * imagpc, int M, int L )
{
    printf("HI1~~~~\n");
    //��Matlab�����ɵ����ݸ���c�У���c���ٸ�GPU///////////////////////////////
    double2 *h_h;
    double2 *h_echo;
    double2 *h_pc;
    h_h=(double2*)malloc(sizeof(double2)*M*L);
    h_echo=(double2*)malloc(sizeof(double2)*L);
    h_pc=(double2*)malloc(sizeof(double2)*M*L);
    for(int i=0; i<M*L;++i)
    {
        h_h[i].x=realh[i];
        h_h[i].y=imagh[i];
        h_echo[i].x=realecho[i];
        h_echo[i].y=imagecho[i];
    }
    printf("HI~~~~\n");
    /*for(int i=0; i<L;++i)
    {
        //printf("h[%d]=%0.8f+%0.8fi\n",i+1,realh[i],imagh[i]);
        printf("h[%d]=%0.8f+%0.8fi\n",i+1,h_h[i].x,h_h[i].y);
    }*/

    //����GPU����/////////////////////////////////////
    hipfftDoubleComplex *dev_h;//���깲�ת�Ĵ��ݺ���
    hipfftDoubleComplex *dev_echo;//�ز�
    hipfftDoubleComplex *dev_pc;//��ѹʱ����
    hipfftDoubleComplex *dev_pcfft;//��ѹƵ����
    //�����Դ�////////////////////////////////////
    hipMalloc(&dev_h, M * L * sizeof(hipfftDoubleComplex));
    hipMalloc(&dev_echo, M * L * sizeof(hipfftDoubleComplex));
    hipMalloc(&dev_pc, M * L * sizeof(hipfftDoubleComplex));
    hipMalloc(&dev_pcfft, M * L * sizeof(hipfftDoubleComplex));
    //���ڴ����ݿ������Դ� //////////////////////////////////
    hipMemcpy(dev_h, h_h, M * L * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(dev_echo, h_echo, M * L * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(dev_pc, h_pc, M * L * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    //����fft�ƻ�
    hipfftHandle plan;
    hipfftPlan1d(&plan,L,HIPFFT_Z2Z,M);
    hipfftExecZ2Z(plan,dev_h,dev_h,HIPFFT_FORWARD);//���ݺ���Ƶ��
    hipfftExecZ2Z(plan,dev_echo,dev_echo,HIPFFT_FORWARD);
    //��ˣ�Ƶ����ѹ
    dim3 BlockPerGrid(1,M);//���������block��ά��
    dim3 threadPerBlock(1,L);//ÿ��block�������߳�ά��
    DianCheng<<<BlockPerGrid,threadPerBlock>>>(dev_h,dev_echo,dev_pcfft,M,L);
    //�渵��Ҷʱ����ѹ���
    hipfftExecZ2Z(plan,dev_pcfft,dev_pc,1);//ifft
    hipMemcpy(h_pc, dev_pc, M * L * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    for(int i=0; i<M*L;++i)
    {
        realpc[i]=h_pc[i].x/200.0;
        imagpc[i]=h_pc[i].y/200.0;
    }
    //realpc=&(h_pc->x);
    //imagpc=&(h_pc->y);
    //�ͷ���Դ
	free(h_h);
	free(h_echo);
	free(h_pc);
	hipFree(dev_h);
	hipFree(dev_echo);
	hipFree(dev_pc);
	hipFree(dev_pcfft);
}

