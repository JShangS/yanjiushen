#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void DianCheng(hipfftDoubleComplex *a,  hipfftDoubleComplex *b, hipfftDoubleComplex *c,int M, int L)//��˵�GPU����
{
      int tx = threadIdx.x;
      int by = blockIdx.y;
      int i=by*L+tx;
      if (i<=M*L)
    {
       c[i].x = a[i].x * b[i].x-a[i].y*b[i].y;
       c[i].y = a[i].x * b[i].y+a[i].y*b[i].x;
    }   
}